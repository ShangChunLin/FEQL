#include "hip/hip_runtime.h"
#include<stdio.h>
#include<cstdlib>
#include<cmath>
#include<iostream>
#include<fstream>
#include <numeric> 

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include<thrust/device_vector.h>
#include<thrust/reduce.h>
#include<thrust/extrema.h>

#include <limits>
#include <cfloat>
#include <ctime>



#define  D2H hipMemcpyDeviceToHost
#define  H2D hipMemcpyHostToDevice
#define  D2D hipMemcpyDeviceToDevice
#define  H2H hipMemcpyHostToHost

using namespace std;

dim3 threads (128);
dim3 blocks  (128);

int avg_timesh = pow(2,26)/threads.x/blocks.x;
int avg_stepsh = pow(2,10);

__constant__ int avg_times;
__constant__ int avg_steps;
__constant__ int intL;
__constant__ float L ,dx, z;
__constant__ float Va1,Vb1,Vc1;
__constant__ float Va2,Vb2,Vc2;
__constant__ float Va3,Vb3,Vc3;
__constant__ float epsilon;

float energy=0;
float zh=0;
const float Lh = 32.0;
const int intLh = int(Lh);
const float dxh = 1.0/32.0;
const int Nh = Lh/dxh;
__constant__ int N;

float densityh;

float Va1h,Vb1h,Vc1h;
float Va2h,Vb2h,Vc2h;
float Va3h,Vb3h,Vc3h;

float epsilonh;
int warm_up_step = pow(10,4);

void ini_parameter(int seed);
void initial_particle(float* particles,int* existh,int seed);
void MC_cpu_warmup(float* particlesh,int* existh,int steps,int seed);
void MC_gpu(float* particles,int* exist,int* density,float* result);
float cal_Vext_cpu (float x);
void display (float*rho,int i);

int main (void)
{   
    hipDeviceReset();
    hipSetDevice(0);
    int batch_size = pow(2,4);
    ofstream fout("MC_parameter.dat");
    ofstream fout2("MC_inform.dat");
    fout<<Lh<<'\t'<<dxh<<'\t'<<batch_size<<endl;
    for(int bat=0;bat<batch_size;bat++){
        ini_parameter(bat*2+1);
        fout2<<bat<<'\t'<<epsilonh<<'\t'<<zh<<endl;
        float *particlesh = new float [intLh];
        int *existh = new int [intLh];
        float *particles,*avg_density_result;
        int *avg_density,*exist;
        hipMalloc((void**)&particles,sizeof(float)*threads.x*blocks.x*intLh);
        hipMalloc((void**)&exist,sizeof(int)*threads.x*blocks.x*intLh);
        hipMalloc((void**)&avg_density,sizeof(int)*threads.x*blocks.x*Nh);
        hipMalloc((void**)&avg_density_result,sizeof(float)*Nh);
        hipMemset(particles,0,sizeof(float)*threads.x*blocks.x*intLh);
        hipMemset(exist,0,sizeof(int)*threads.x*blocks.x*intLh);
        hipMemset(avg_density,0,sizeof(int)*threads.x*blocks.x*Nh);
        hipMemset(avg_density_result,0,sizeof(float)*Nh);

        initial_particle(particlesh,existh,bat+1);
        MC_cpu_warmup(particlesh,existh,warm_up_step,bat+1);

        for(int i =0;i<threads.x*blocks.x;i++)
        {
            MC_cpu_warmup(particlesh,existh,128,i*batch_size+bat);
            hipMemcpy(&particles[i*intLh],particlesh,sizeof(float)*intLh,H2D);
            hipMemcpy(&exist[i*intLh],existh,sizeof(float)*intLh,H2D);
        }

        cout<<"initial done"<<endl;
        cout<<"total samples"<<'\t'<<avg_timesh*blocks.x*threads.x<<endl; 
        MC_gpu(particles,exist,avg_density,avg_density_result);
        cout<<"gpu done"<<endl;
        float *resulth= new float [Nh];
        hipMemcpy(resulth,avg_density_result,sizeof(float)*Nh,D2H);
        display(resulth,bat);
        free(particlesh);
        free(resulth);
        free(existh);
        hipFree(exist);
        hipFree(particles);
        hipFree(avg_density);
        hipFree(avg_density_result);

    }
    return 0;
}


void display (float*rho,int bat)
{
    char* s = new char[100];
    sprintf(s,"rho_%d.dat",bat);
    char* s2 = new char[100];
    sprintf(s2,"Vext_%d.dat",bat);
    ofstream fout (s);
    ofstream fout_V (s2);
    for(int i=0;i<Nh;i++){
        fout<<rho[i]/threads.x/blocks.x/avg_timesh<<endl;
        fout_V<<cal_Vext_cpu(i*dxh)<<endl;
    }
    free(s);
    free(s2);
}


void ini_parameter(int seed)
{
    hipMemcpyToSymbol(HIP_SYMBOL(avg_times), &avg_timesh, sizeof(avg_timesh));
    hipMemcpyToSymbol(HIP_SYMBOL(avg_steps), &avg_stepsh, sizeof(avg_stepsh));
    
    hipMemcpyToSymbol(HIP_SYMBOL(N), &Nh, sizeof(Nh));
    hipMemcpyToSymbol(HIP_SYMBOL(dx), &dxh, sizeof(dxh));
    hipMemcpyToSymbol(HIP_SYMBOL(L), &Lh, sizeof(Lh));
    cout<<"seed"<<'\t'<<seed<<endl; 
    srand(seed); 
    //cout<<"test"<<'\t'<<rand()<<endl; 
    
    Va1h = (float)rand()/(float)(RAND_MAX)*8-4;//-3:3
    Vb1h = (float)rand()/(float)(RAND_MAX)*Lh/2+Lh/4; 
    Vc1h = (float)rand()/(float)(RAND_MAX/2)+1;
    Va2h = (float)rand()/(float)(RAND_MAX)*8-4;//-3:3
    Vb2h = (float)rand()/(float)(RAND_MAX)*Lh/2+Lh/4; 
    Vc2h = (float)rand()/(float)(RAND_MAX/2)+1;
    Va3h = (float)rand()/(float)(RAND_MAX)*8-4;//-3:3
    Vb3h = (float)rand()/(float)(RAND_MAX)*Lh/2+Lh/4; 
    Vc3h = (float)rand()/(float)(RAND_MAX/2)+1;

    epsilonh =  (float)rand()/(float)(RAND_MAX)*4+2; //0.5-1.5
    zh =  (float)rand()/(float)(RAND_MAX)*2+1; //0-3 exp(0)=1

    /**/
    //zh = 3;
    //epsilonh=1.5;
    //Vah = 1;
    //Vbh = 8;
    //Vch = 3;
    /**/
    
    //cout<<Vah<<'\t'<<Vbh<<'\t'<<Vch<<endl;

    hipMemcpyToSymbol(HIP_SYMBOL(epsilon), &epsilonh, sizeof(Lh));
    hipMemcpyToSymbol(HIP_SYMBOL(z), &zh, sizeof(zh));
    hipMemcpyToSymbol(HIP_SYMBOL(intL), &intLh, sizeof(intLh));

    
    hipMemcpyToSymbol(HIP_SYMBOL(Va1), &Va1h, sizeof(Va1h));
    hipMemcpyToSymbol(HIP_SYMBOL(Vb1), &Vb1h, sizeof(Vb1h));
    hipMemcpyToSymbol(HIP_SYMBOL(Vc1), &Vc1h, sizeof(Vc1h));
    hipMemcpyToSymbol(HIP_SYMBOL(Va2), &Va2h, sizeof(Va2h));
    hipMemcpyToSymbol(HIP_SYMBOL(Vb2), &Vb2h, sizeof(Vb2h));
    hipMemcpyToSymbol(HIP_SYMBOL(Vc2), &Vc2h, sizeof(Vc2h));
    hipMemcpyToSymbol(HIP_SYMBOL(Va3), &Va3h, sizeof(Va3h));
    hipMemcpyToSymbol(HIP_SYMBOL(Vb3), &Vb3h, sizeof(Vb3h));
    hipMemcpyToSymbol(HIP_SYMBOL(Vc3), &Vc3h, sizeof(Vc3h));
    
}

void initial_particle(float* particles,int* exist,int seed)
{
    srand(seed);
    for(int i=0;i<intLh;i++){
        float x = (float)rand()/(float)(RAND_MAX)*Lh;
        particles[i]=x;
        x = (float)rand()/(float)(RAND_MAX)-0.5;
        exist[i]=0;
        //cout<<i<<'\t'<<particles[i]<<endl;
    }


}

__device__ __host__ float Uij(float dissq)
{
    return 4*(powf(1.0/dissq,6)-powf(1.0/dissq,3));
}


__device__ float cal_Vext_gpu (float x)
{
    float V = 0;
    V+=-Va1*expf(-powf((x-Vb1),2)/(2*Vc1*Vc1));
    V+=-Va2*expf(-powf((x-Vb2),2)/(2*Vc2*Vc2));
    V+=-Va3*expf(-powf((x-Vb3),2)/(2*Vc3*Vc3));
    if(x<=1)return pow(10.0,8);
    return V;
}

__device__ float dissq_gpu(float x1,float x2)
{
    float Dx=fabsf(x1-x2);
    if(Dx>L/2)Dx-=L;
    return Dx*Dx;
}


__device__ float particle_energy_gpu(float this_particle ,int* exist,float* particles,int p)
{
    float energy=0;
    for(int j=0;j<intL;j++){
        if(p!=j&&exist[j]==1){
            float dissq = dissq_gpu(this_particle,particles[j]);
            if(dissq<=1)return 100000000;
            energy+=Uij(dissq)*epsilon;
        }
    }
    
    energy+=cal_Vext_gpu(this_particle);

    return energy;
}

__device__ int sum_tot(int* a)
{
    int b=0;
    for(int i=0;i<intL;i++)b+=a[i];
    return b;
}

__device__ void cum_sum(int* a,int *b)
{ 
    for(int i=0;i<intL;i++)a[i]=b[i];
    for(int i=1;i<intL;i++)a[i]+=a[i-1];
}

__device__ void deletion_gpu(float* particle,int* exist,int*cum_exist,int MM,int Np,float dice)
{
        //int Np = sum_tot(exist);//current particle number
        //cum_sum(cum_exist,exist);
        //int M=hiprand(&state)%(Np)+1;//choose Mth exist particle (neccesary)
        for(int i=0;i<intL;i++)
        {
            if(exist[i]==1 && cum_exist[i]==MM)
            //if(exist[i]==1)
            {
                float ene = -particle_energy_gpu(particle[i],exist,particle,i);
                float prop = 1.0/L*Np/z*expf(-ene);
                if(prop>1)exist[i]=0;
                else{
                    //float dice = (hiprand_uniform(&state));
                    if(dice<prop)exist[i]=0;
                }
                break;
            }
        }
}

__device__ void insertion_gpu(float* particle,int* exist,int*cum_exist,float varL,int Np,float dice)
{
        //int Np = sum_tot(exist);
        for(int i=0;i<intL;i++)
        {
            if(exist[i]==0)
            {
                particle[i]=varL;
                //printf("particle[i]=%f\n",particle[i]);
                float ene = particle_energy_gpu(particle[i],exist,particle,i);
                float prop = z*L/(Np+1)*expf(-ene);
                if(prop>1)exist[i]=1;
                else{
                    //float dice = (hiprand_uniform(&state));
                    if(dice<prop)exist[i]=1;
                }
                break;
            }
        }
}

// each thread hold one sub system
__global__ void MC_gpu_kernel(float* particles,int* exist,int*cum_exist,int* avg_density)
{
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx>=gridDim.x*blockDim.x){
        printf("out!\n");
        return;
    }
    
    hiprandState_t state;
    hiprand_init(idx,idx*idx,idx*idx*idx,&state);
    for(int iter=0;iter<avg_times;iter++){
        for(int i=0;i<avg_steps;i++){
            float prop = (hiprand_uniform(&state)-0.5);
            int Np = sum_tot(&exist[idx*intL]);//current particle number
            int M=idx*intL;
            float dice = (hiprand_uniform(&state));
            if(prop>0 && Np>0 ){
                cum_sum(&cum_exist[M],&exist[M]);
                int MM=hiprand(&state)%(Np)+1;//choose Mth exist particle (neccesary)
                deletion_gpu(&particles[M],&exist[M],&cum_exist[M],MM,Np,dice);
            }
            if(prop<0 && Np<intL){
                float varL =(hiprand_uniform(&state))*L;
                insertion_gpu(&particles[M],&exist[M],&cum_exist[M],varL,Np,dice);
            }
        }

        for(int i=0;i<intL;i++){
            int position = __float2int_rn(particles[idx*intL+i]/dx);
            position = position%N;
            //printf("position=%d,x/dr=%f",position,particles[idx*intL+i]/dx);
            if(exist[idx*intL+i]==1)avg_density[idx*N+position]+=1;
        }
    }
    //printf("idx=%d,avg_density=%d\n",idx,avg_density[idx*N+N/2]);
}


//sum over column
__global__ void sum_density(int* density,float* result,int num_systems)
{
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx>=N)return;
    for(int j=1;j<num_systems;j++)
    {
        density[idx]+=density[j*N+idx];
    }
        //printf("i=%d,density=%d\n",i,density[i]);
    result[idx]= density[idx]/dx;
}

void MC_gpu(float* particles,int* exist,int* avg_density,float* result)
{
    int* cum_exist;
    hipMalloc((void**)&cum_exist,sizeof(int)*threads.x*blocks.x*intLh);
    MC_gpu_kernel<<<blocks,threads>>>(particles,exist,cum_exist,avg_density);
    hipDeviceSynchronize();  
    cout<<"gpu done"<<endl;
    sum_density<<<(Nh+threads.x-1)/threads.x,threads>>>(avg_density,result,blocks.x*threads.x);
    hipDeviceSynchronize();  
    cout<<"avg done"<<endl;
    hipFree(cum_exist);
    
}


float cal_Vext_cpu (float x)
{
    float V = 0;
    
    V+=-Va1h*expf(-powf((x-Vb1h),2)/(2*Vc1h*Vc1h));
    V+=-Va2h*expf(-powf((x-Vb2h),2)/(2*Vc2h*Vc2h));
    V+=-Va3h*expf(-powf((x-Vb3h),2)/(2*Vc3h*Vc3h));
    if(x<=1.0)V=pow(10,8); 
    return V;
}

float dissq_cpu(float x1,float x2)
{
    float Dx=fabs(x1-x2);
    if(Dx>Lh/2)Dx-=Lh;
    return Dx*Dx;
}


float energy_cpu(float* particles,int *exist)
{
    float energy=0;
    
    for(int i=0;i<intLh;i++){
        for(int j=0;j<intLh;j++){
            if(i!=j && exist[i]*exist[j]==1){
                float dissq = dissq_cpu(particles[i],particles[j]);
                energy+=Uij(dissq)/2*epsilonh;
                if(dissq<1)cout<<"overlapped!!!"<<endl;
            }
        }
    }
    
    for(int i=0;i<intLh;i++){
        if(exist[i]==1)energy+=cal_Vext_cpu(particles[i]);
    }
    return energy;
}

float particle_energy_cpu(float this_particle ,int* exist,float* particles,int p)
{
    float energy=0;
    
    for(int j=0;j<intLh;j++){
        if(p!=j && exist[j]==1 ){
            float dissq = dissq_cpu(this_particle,particles[j]);
            if(dissq<1) return pow(10,8);
            energy+=Uij(dissq)*epsilonh;
        }
    }
    
    energy+=cal_Vext_cpu(this_particle);

    return energy;
}


void insertionh(float* particles,int* exist)
{
    int N = accumulate(exist,exist+intLh,0);      
    for(int i=0;i<intLh;i++)
    {
        if(exist[i]==0)
        {
            particles[i]=(float)rand()/(float)(RAND_MAX)*Lh;
            double ene = particle_energy_cpu(particles[i] ,exist,particles,i);
            double prop = zh*Lh/(N+1)*exp(-ene);
            if(prop>=1)exist[i]=1;
            else{
                float dice = (float)rand()/(float)(RAND_MAX);
                if(dice<prop)exist[i]=1;
            }
            break;
        }
    }
}

void deletionh(float* particles,int* exist)
{
    int *cum_exist = new int[intLh];
    partial_sum (exist, exist+intLh, cum_exist);
    int N = accumulate(exist,exist+intLh,0);
    int M = (int)rand()%(N)+1;

    for(int i=0;i<intLh;i++)
    {
        if(exist[i]==1 && cum_exist[i] == M)
        {
            double ene = -particle_energy_cpu(particles[i] ,exist,particles,i);
            double prop = 1.0*N/Lh/zh*exp(-ene);
            if(prop>=1)exist[i]=0;
            else{
                float dice = (float)rand()/(float)(RAND_MAX);
                if(dice<prop){
                    exist[i]=0;
                }
            }
            break;
        }
    }
    free(cum_exist);
}



void MC_cpu_warmup(float* particles,int* exist,int steps,int seed)
{
    //ofstream fout ("warmup.dat");
    srand(seed+1);
    //cout<<energy<<endl;
    int sum_N=0;
    int count=0;
    for(int i =0;i<steps;i++){
        srand(i);
        float prop = (float)rand()/(float)(RAND_MAX)-0.5;
        int N =accumulate(exist,exist+intLh,0);      
        if(prop>0 && N<intLh)insertionh(particles,exist);
        else if(prop<0 && N>0)deletionh(particles,exist);
        N =accumulate(exist,exist+intLh,0); 
        energy = energy_cpu(particles,exist);
        if(i>1000){
            sum_N+=N;
            count++;
        }
        //cout<<i<<'\t'<<N<<'\t'<<energy<<endl;
    }
    //cout<<sum_N/count/Lh<<endl;
}


